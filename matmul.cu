/*
module load intel CUDA
compile
nvcc -ccbin=icc -o matmul.exe matmul.cu
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#include <assert.h>
#include <chrono>

#define MAXINT 2048
enum QUARTER {
    m11,
    m12,
    m21,
    m22
};

static void calculate_block_thread_dim(int k, int& block_dim, int& thread_dim) {
    // block size should be in multiple of 32
    // make num threads to be 256 = 16*16
    int matrix_dim = 1 << k;
    block_dim = matrix_dim/16<1? 1 : matrix_dim/16;
    thread_dim = matrix_dim / block_dim;
    
    // printf("------------------");
    // printf("matrix_dim %d\n", matrix_dim);
    // printf("block_dim %d\n", block_dim);
    // printf("thread_dim %d\n", thread_dim);
    // printf("num_blocks %d\n", block_dim*block_dim);
    // printf("num_threads %d\n", thread_dim*thread_dim);
}

int **dM1, **dM2, **dM3, **dM4, **dM5, **dM6, **dM7;
int *tmp;
int block_dim;

__device__
static void get_sub_offset(int curr_dim, QUARTER q, int& x_offset, int& y_offset) {
    int new_dim = curr_dim/2;
    switch (q) {
        case m11:
            // do nothing
            break;
        case m12:
            y_offset = new_dim;
            break;
        case m21:
            x_offset = new_dim;
            break;
        case m22:
            x_offset = new_dim;
            y_offset = new_dim;
            break;
        default:
            assert(false);
    }
}

__global__
void matAdd(int* A, int* B, int* C, int org_dim, QUARTER qA, QUARTER qB, QUARTER qC, int k, bool is_subtract=false)
{   
    int curr_dim = 1 << k;
    int new_dim = curr_dim /2;
    int Ax_offset = 0;
    int Ay_offset = 0;
    int Bx_offset = 0;
    int By_offset = 0;
    int Cx_offset = 0;
    int Cy_offset = 0;

    int thread_posx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_posy = blockIdx.y * blockDim.y + threadIdx.y;

    get_sub_offset(curr_dim, qA, Ax_offset, Ay_offset);
    get_sub_offset(curr_dim, qB, Bx_offset, By_offset);
    get_sub_offset(curr_dim, qC, Cx_offset, Cy_offset);

    if (is_subtract) {
        C[(Cx_offset+thread_posx)*org_dim + Cy_offset + thread_posy] = A[(Ax_offset+thread_posx)*org_dim + Ay_offset + thread_posy] - B[(Bx_offset+thread_posx)*org_dim + By_offset + thread_posy];
        return;
    }
    C[(Cx_offset+thread_posx)*org_dim + Cy_offset + thread_posy] = A[(Ax_offset+thread_posx)*org_dim + Ay_offset + thread_posy] + B[(Bx_offset+thread_posx)*org_dim + By_offset + thread_posy];
}

__global__
void matCopy(int* fromM, int* toM, int org_dim, QUARTER fromQ, QUARTER toQ, int k) {
    int curr_dim = 1 << k;
    int new_dim = curr_dim /2;
    int fromx_offset = 0;
    int fromy_offset = 0;
    int tox_offset = 0;
    int toy_offset = 0;

    int thread_posx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_posy = blockIdx.y * blockDim.y + threadIdx.y;

    get_sub_offset(curr_dim, fromQ, fromx_offset, fromy_offset);
    get_sub_offset(curr_dim, toQ, tox_offset, toy_offset);
    toM[(tox_offset+thread_posx)*org_dim + toy_offset + thread_posy] = fromM[(fromx_offset+thread_posx)*org_dim + fromy_offset + thread_posy];
}

static void normalMatMul(int*A, int*B, int*C, int dim) {
    for (int i = 0; i < dim; i ++) {
        for (int j = 0; j < dim; j++) {
            // for the current row of A and col of B
            C[i*dim + j] = 0;
            for (int k = 0; k < dim; k ++) {
                C[i*dim + j] += A[i*dim+k] * B[k*dim + j];
            }
        }
    }
}

// __global__
// void matMul(int* A, int* B, int* C, int curr_dim) {
//     assert(curr_dim == 1);
//     if (threadIdx.x == 0 && threadIdx.y == 0) {
//         C[0] = A[0] * B[0];
//     }
// }

__global__
void normalParallelMatMulKernel(int* A, int* B, int* C, int org_dim, int k) {
    // each thread computes an element of C
    int curr_dim = 1 << k;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    for (int i = 0; i < curr_dim; i++) {
        sum += A[row*org_dim + i] * B[i*org_dim + col];
    }
    C[row*org_dim + col] = sum;
}

static void printMat(int* mat, int xDim, int yDim, char* name) {
    printf("Matrix %s:\n", name);
    for (int i = 0; i < xDim; i++) {

        for(int j = 0; j < yDim; j++) {
            printf("\t%d", mat[i*xDim+j]);
        }
        printf("\n");
    }
}

static void printDevMat(int* mat, int xDim, int yDim, char* name) {
    hipMemcpy(tmp, mat, xDim*yDim*sizeof(int), hipMemcpyDeviceToHost);
    printMat(tmp, xDim, yDim, name);
}

void normalParallelMatMul(int* A, int* B, int* C, int org_dim, int k) {
    int curr_dim = 1 << k;

    int block_dim = 0;
    int thread_dim = 0;

    calculate_block_thread_dim(k, block_dim, thread_dim);

    assert(block_dim != 0);
    assert(thread_dim != 0);
    assert(thread_dim * block_dim == curr_dim);

    dim3 blocks(block_dim, block_dim);
    dim3 grid(thread_dim, thread_dim);
    normalParallelMatMulKernel<<<blocks, grid>>>(A, B, C, org_dim, k);
}



void Strassen(int** dA, int** dB, int** dC, int org_dim, int k, int k_lim) {
    int curr_dim = 1 << k;
    int new_dim = curr_dim /2;
    // matAdd<<<1, grid>>>(dA[k], dB[k], dC[k], org_dim, m11, m11);

    if (k == k_lim) {
        // assert(curr_dim == 1);
        // matMul<<<1, 1>>>(dA[k], dB[k], dC[k], curr_dim);
        normalParallelMatMul(dA[k], dB[k], dC[k], org_dim, k);
        return;
    }
    int block_dim = 0;
    int thread_dim = 0;

    calculate_block_thread_dim(k-1, block_dim, thread_dim);

    assert(block_dim != 0);
    assert(thread_dim != 0);
    assert(thread_dim * block_dim == new_dim);

    dim3 blocks(block_dim, block_dim);
    dim3 grid(thread_dim, thread_dim);
    // M1v
    matAdd<<<blocks, grid>>>(dA[k], dA[k], dA[k-1], org_dim, m11, m22, m11, k);
    matAdd<<<blocks, grid>>>(dB[k], dB[k], dB[k-1], org_dim, m11, m22, m11, k);
    Strassen(dA, dB, dM1, org_dim, k-1, k_lim);

    // M2v
    matAdd<<<blocks, grid>>>(dA[k], dA[k], dA[k-1], org_dim, m21, m22, m11, k);
    matCopy<<<blocks, grid>>>(dB[k], dB[k-1], org_dim, m11, m11, k);
    Strassen(dA, dB, dM2, org_dim, k-1, k_lim);

    // M3v
    matCopy<<<blocks, grid>>>(dA[k], dA[k-1], org_dim, m11, m11, k);
    matAdd<<<blocks, grid>>>(dB[k], dB[k], dB[k-1], org_dim, m12, m22, m11, k, true);
    Strassen(dA, dB, dM3, org_dim, k-1, k_lim);

    // M4v
    matCopy<<<blocks, grid>>>(dA[k], dA[k-1], org_dim, m22, m11, k);
    matAdd<<<blocks, grid>>>(dB[k], dB[k], dB[k-1], org_dim, m21, m11, m11, k, true);
    Strassen(dA, dB, dM4, org_dim, k-1, k_lim);

    // M5v
    matAdd<<<blocks, grid>>>(dA[k], dA[k], dA[k-1], org_dim, m11, m12, m11, k);
    matCopy<<<blocks, grid>>>(dB[k], dB[k-1], org_dim, m22, m11, k);
    Strassen(dA, dB, dM5, org_dim, k-1, k_lim);

    // M6v
    matAdd<<<blocks, grid>>>(dA[k], dA[k], dA[k-1], org_dim, m21, m11, m11, k, true);
    matAdd<<<blocks, grid>>>(dB[k], dB[k], dB[k-1], org_dim, m11, m12, m11, k);
    Strassen(dA, dB, dM6, org_dim, k-1, k_lim);

    // M7
    matAdd<<<blocks, grid>>>(dA[k], dA[k], dA[k-1], org_dim, m12, m22, m11, k, true);
    matAdd<<<blocks, grid>>>(dB[k], dB[k], dB[k-1], org_dim, m21, m22, m11, k);
    Strassen(dA, dB, dM7, org_dim, k-1, k_lim);




    // C11
    matAdd<<<blocks, grid>>>(dM1[k-1], dM4[k-1], dC[k], org_dim, m11, m11, m11, k);
    matAdd<<<blocks, grid>>>(dC[k], dM5[k-1], dC[k], org_dim, m11, m11, m11, k, true);
    matAdd<<<blocks, grid>>>(dC[k], dM7[k-1], dC[k], org_dim, m11, m11, m11, k);

    // C12
    matAdd<<<blocks, grid>>>(dM3[k-1], dM5[k-1], dC[k], org_dim, m11, m11, m12, k);

    // C21
    matAdd<<<blocks, grid>>>(dM2[k-1], dM4[k-1], dC[k], org_dim, m11, m11, m21, k);

    // C22
    matAdd<<<blocks, grid>>>(dM1[k-1], dM2[k-1], dC[k], org_dim, m11, m11, m22, k, true);
    matAdd<<<blocks, grid>>>(dC[k], dM3[k-1], dC[k], org_dim, m22, m11, m22, k);
    matAdd<<<blocks, grid>>>(dC[k], dM6[k-1], dC[k], org_dim, m22, m11, m22, k);

}




int main(int argc, char** argv)
{
    srand (time(NULL));
    if (argc != 3) {
        printf("Usage: %s <k> <k_prime>\n", argv[0]);
        exit(0);
    }


    /*
    Strassen recur for k levels
    Perform normal matmul after k_prime levels
    */
    int k   = atoi(argv[1]);
    int k_prime  = atoi(argv[2]);
    int k_lim = k - k_prime;
    // int block_dim_exp = atoi(argv[3]);

    if (k_prime >= k || k_prime < 1) {
        printf("k > k_prime > 1\n");
        exit(0);
    }

    // if (block_dim_exp >= k || block_dim_exp < 1) {
    //     printf("k > block_dim_exp > 1\n");
    //     exit(0);
    // }

    int n = 1 << k;
    int size = n*n;
    int bytes = size*sizeof(int);
    // block_dim = 1 << block_dim_exp;
    // assert(block_dim <= n);

    int *hA, *hB, *hC, *testC;
    int **dA, **dB, **dC;
    hA = (int*)malloc(bytes);
    hB = (int*)malloc(bytes);
    hC = (int*)malloc(bytes);
    testC = (int*)malloc(bytes);
    tmp = (int*)malloc(bytes);

    // init host matrices

    for (int i = 0; i < size; i++) {
        hA[i] = rand() % MAXINT;
        hB[i] = rand() % MAXINT;
        hC[i] = 0;
        testC[i] = 0;
        tmp[i] = 0;
    }

    // alloc device matrices

    dA = (int**)malloc((k+1)*sizeof(int*));
    dB = (int**)malloc((k+1)*sizeof(int*));
    dC = (int**)malloc((k+1)*sizeof(int*));
    for (int i = 0; i < k+1; i++) {
        hipMalloc(&dA[i], bytes);
        hipMalloc(&dB[i], bytes);
        hipMalloc(&dC[i], bytes);
    }

    hipMemcpy(dA[k], hA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dB[k], hB, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dC[k], hC, bytes, hipMemcpyHostToDevice);

    for (int i = 0; i < k; i++) {
        hipMemcpy(dA[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dB[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dC[i], hC, bytes, hipMemcpyHostToDevice);
    }

    // alloc temp matrices
    dM1 = (int**)malloc((k+1)*sizeof(int*));
    dM2 = (int**)malloc((k+1)*sizeof(int*));
    dM3 = (int**)malloc((k+1)*sizeof(int*));
    dM4 = (int**)malloc((k+1)*sizeof(int*));
    dM5 = (int**)malloc((k+1)*sizeof(int*));
    dM6 = (int**)malloc((k+1)*sizeof(int*));
    dM7 = (int**)malloc((k+1)*sizeof(int*));
    for (int i = 0; i < k+1; i++) {
        hipMalloc(&dM1[i], bytes);
        hipMalloc(&dM2[i], bytes);
        hipMalloc(&dM3[i], bytes);
        hipMalloc(&dM4[i], bytes);
        hipMalloc(&dM5[i], bytes);
        hipMalloc(&dM6[i], bytes);
        hipMalloc(&dM7[i], bytes);

        hipMemcpy(dM1[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dM2[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dM3[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dM4[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dM5[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dM6[i], hC, bytes, hipMemcpyHostToDevice);
        hipMemcpy(dM7[i], hC, bytes, hipMemcpyHostToDevice);
    }

    // printMat(hA, n, n, "A");
    // printMat(hB, n, n, "B");
    // printMat(hC, n, n, "init C");
    auto start = std::chrono::system_clock::now();
    Strassen(dA, dB, dC, n, k, k_lim);
    auto end = std::chrono::system_clock::now();
    printf("Strassen took %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    // normalParallelMatMul(dA[k], dB[k], dC[k], n, k);

    // int new_dim = n /2;
    // dim3 grid(new_dim, new_dim);
    // matAdd<<<1, grid>>>(dA[k], dB[k], dC[k], n, m11, m12, m12, k, true);
    // matCopy<<<1, grid>>>(dC[k], dA[k], n, m12, m21, k);

    hipMemcpy(hC, dC[k], bytes, hipMemcpyDeviceToHost);
    // printMat(hC, n, n, "result C");

    start = std::chrono::system_clock::now();
    normalMatMul(hA, hB, testC, n);
    end = std::chrono::system_clock::now();
    printf("Single sequential took %lldms\n", std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

    // printMat(testC, n, n, "test C");

    int err_count = 0;
    for (int i = 0; i < size; i++) {
        if (hC[i] != testC[i]) err_count ++;
    }
    if (err_count == 0) {
        printf("Congrats, matmul calculated correctly!\n");
    } else {
        printf("Houston, we have a problem! Err_count = %d\n", err_count);
    }

    for (int i = 0; i < k+1; i++) {
        hipFree(dA[i]);
        hipFree(dB[i]);
        hipFree(dC[i]);

        hipFree(dM1[i]);
        hipFree(dM2[i]);
        hipFree(dM3[i]);
        hipFree(dM4[i]);
        hipFree(dM5[i]);
        hipFree(dM6[i]);
        hipFree(dM7[i]);
    }

    free(dM1);
    free(dM2);
    free(dM3);
    free(dM4);
    free(dM5);
    free(dM6);
    free(dM7);
    free(dA);
    free(dB);
    free(dC);
    free(hA);
    free(hB);
    free(hC);
    free(tmp);
    free(testC);
}
